#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>

#define RANGE 1048576

template<typename T>
void create_random_list(T* data, int size);

struct IntegerComparator {
    __host__ __device__ bool operator()(int a, int b) const {
        return a*a-1 <= b*b+2;
    }
};

template<typename T>
void create_random_list(T* data, int size) {
  long temp;
//printf("size:%d\n", size);
  for(int i=0; i<size; i++) {
//    data[i].key = rand()%RANGE + min;
//    data[i].val = rand()%RANGE + min;
    data[i] = (rand()%RANGE);
    temp = rand()%RANGE;
    data[i] += (temp<<32);
  }
}

int main(int argc, char ** argv) {
    hipEvent_t start, stop;
    float time_elapsed = 0.0;

    if (argc != 2) {
      printf("Usage: ./thrust <N>\n");
    }
    int N = atoi(argv[1]);
    // Host vector of integers
    int* h_array = (int*) malloc(N * sizeof(int));

    create_random_list<int>(h_array, N);

    // Copy the host array to device
    thrust::device_vector<int> d_vec(h_array, h_array + N);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Sort on device using the custom comparator
    thrust::sort(d_vec.begin(), d_vec.end(), IntegerComparator());

    hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_elapsed, start, stop);

  printf("%f\n", time_elapsed);

    // Copy the sorted device vector back to host
    // thrust::copy(d_vec.begin(), d_vec.end(), h_array);
    

    return 0;
}