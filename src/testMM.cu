#include "hip/hip_runtime.h"
/*
 * (C) Copyright 2016-2018 Ben Karsin, Nodari Sitchinava
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <random>
#include <algorithm>
#include "multimergesort.hxx"
#include "buildData.h"

#include <stdlib.h>

#define DEBUG 1 // Set this to 1 to check that the output is correctly sorted
#define PRINT 0 // Set this to 1 to print first M elements of the array for further debugging
#define ITERS 1 // Number of iterations to compute average runtime
#define BLOCKS 128

/* CPU FUNCTION HEADERS*/
template <typename T>
void test_multimergesort(int p, int N);
template <typename T>
void test_squareSort(int N);
template <typename T>
void CPUsort(T *arr, int N);
template <typename T, fptr_t f>
void test_arrayEquality(T *arr1, T *arr2, int N);
template <typename T, fptr_t f>
void selection_sort(T* a, int N);

int main(int argc, char **argv)
{

  if (argc != 2){
    printf("Usage: ./testMM <N>\n");
    exit(1);
  }

  int N = atoi(argv[1]);
  // test_multimergesort<DATATYPE>(BLOCKS, N);
  test_squareSort<DATATYPE>(N);

  return 0;
}

// Create random data and sort it...
template <typename T>
void test_multimergesort(int p, int N)
{

  hipEvent_t start, stop;
  float time_elapsed = 0.0;
  float minTime = 99999;
  float maxTime = 0.0;

  // Create sample sorted lists
  T *h_data = (T *)malloc(N * sizeof(T));

  T *d_data;
  T *d_output;
  hipMalloc(&d_data, N * sizeof(T));
  hipMalloc(&d_output, N * sizeof(T));
  float total_time = 0.0;

  srand(0); // time(NULL)

  for (int it = 0; it < ITERS; it++)
  {

    // Create random list to be sorted
    create_random_list<T>(h_data, N, 0);

    // Copy list to GPU
    hipMemcpy(d_data, h_data, N * sizeof(T), hipMemcpyHostToDevice);

    // Zero out result array
    hipMalloc(&d_output, N * sizeof(T));
    //  hipMemset(&d_output, 0, N*sizeof(T));

    hipDeviceSynchronize();
    // Timer functions
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Run GPU-MMS.  T is datatype and cmp is comparison function (defined in cmp.hxx)
    d_output = multimergesort<T, cmp>(d_data, d_output, h_data, p, N);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);
    total_time += time_elapsed;
    if (time_elapsed < minTime)
      minTime = time_elapsed;
    if (time_elapsed > maxTime)
      maxTime = time_elapsed;
    time_elapsed = 0.0;

    if (it < ITERS - 1)
      hipFree(&d_output);
  }
  total_time = total_time / ITERS;
  printf("%lf %lf %lf\n", total_time, minTime, maxTime);

  // copy sorted result back to CPU
  hipMemcpy(h_data, d_output, N * sizeof(T), hipMemcpyDeviceToHost);

// If debug mode is on, check that output is correct
#ifdef DEBUG
  bool error = false;
  int erroneous_index;
  for (int i = 2; i < N - 1; i++)
  {
    if (host_cmp<int>(h_data[i], h_data[i - 1]))
    {
      error = true;
      erroneous_index = i;
      break;
    }
  }
  if (error)
    printf("NOT SORTED! Item at index %d is less than its predecessor.\n", erroneous_index);
  else
    printf("SORTED!\n");

  int greatest_power_of_K = 1024;
  bool error_with_subarrays = false;
  int erroneous_index_subarrays = -1;
  if (greatest_power_of_K * K <= N)
    greatest_power_of_K <<= 2; // where 2 is log(K)
  for (int i=0; i < N; i += greatest_power_of_K){
    for (int j=1; j < greatest_power_of_K; j++){
      if (i + j < N){
        if (host_cmp<int>(h_data[i+j], h_data[i+j-1]))
          error_with_subarrays = true;
        erroneous_index_subarrays = i+j;
        break;
      }
    }
  }
  if (error_with_subarrays)
    printf("NOT SORTED! Item at index %d is less than its predecessor.\n", erroneous_index_subarrays);
  else
    printf("SORTED SUBARRAYS!\n");

#if PRINT == 1
  printf("[%d", h_data[0]);
  for (int i = 1; i < M; i++)
    printf(", %d", h_data[i]);
  printf("]\n");
#endif
// drawBarGraph(h_data, N);
#endif

  hipFree(d_data);
  hipFree(d_output);
  free(h_data);
}

// Function to test just the basecase method
template <typename T>
void test_squareSort(int N){
  hipEvent_t start, stop;
  float time_elapsed = 0.0;
  float total_time = 0.0;

  T *h_data = (T *)malloc(N * sizeof(T));
  T *cpu_data = (T *)malloc(N * sizeof(T));
  T *d_data;
  hipMalloc(&d_data, N * sizeof(T));
  srand(time(NULL));

  for (int it = 0; it < ITERS; it++)
  {
    create_random_list<T>(h_data, N, 0);

    hipMemcpy(d_data, h_data, N * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(cpu_data, h_data, N * sizeof(T), hipMemcpyHostToHost);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    squareSort<T, cmp><<<(((N+M-1)/M) / (THREADS/W)), THREADS>>>(d_data, N); // number of blocks was initially BLOCKS, but that was incorrect. squareSort is implemented to have grid dimensions that scale with input size

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);

    total_time += time_elapsed;
  }

  printf("%lf ", total_time / ITERS);
  printf("\n");
  hipError_t err = hipGetLastError();
  if (err)
    printf("%s\n", hipGetErrorString(err));

  hipMemcpy(h_data, d_data, N * sizeof(T), hipMemcpyDeviceToHost);
  hipFree(d_data);
#ifdef DEBUG
  bool sorted = true;

  int start_of_base_case;
  for (start_of_base_case=0; start_of_base_case<=N-M; start_of_base_case += M)
    selection_sort<int, host_cmp>(cpu_data + start_of_base_case, M);
  selection_sort<int, host_cmp>(cpu_data + start_of_base_case, N-start_of_base_case);
  test_arrayEquality<int, host_cmp>(cpu_data, h_data, M*(N/M)); // Compare a number of elements equal to the greatest multiple of M.
#if PRINT == 1
  printf("[%d", h_data[0]);
  for (int i = 1; i < N; i++)
    printf(", %d", h_data[i]);
  printf("]\n");
#endif

  for (int j = 0; j < N; j += M){
    for (int i = 1; i < M; i++){
      if (i + j < N && host_cmp<int>(h_data[i + j], h_data[j + i - 1]))
        sorted = false;
    }
  }
  if (!sorted)
    printf("NOT SORTED\n");
  else
    printf("SORTED!\n");
  /*
  For drawing a bar graph representing the data

  SDL_Window* window = nullptr;
  SDL_Renderer* renderer = nullptr;

  if (SDL_Init(SDL_INIT_VIDEO) < 0) {
    std::cerr << "SDL could not initialize! SDL_Error: " << SDL_GetError() << std::endl;
    return -1;
  }
  window = SDL_CreateWindow("Bar Graph", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
  if (window == nullptr) {
    std::cerr << "Window could not be created! SDL_Error: " << SDL_GetError() << std::endl;
    return -1;
  }

  renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);
  if (renderer == nullptr) {
    std::cerr << "Renderer could not be created! SDL_Error: " << SDL_GetError() << std::endl;
    return -1;
  }

  bool quit = false;
  SDL_Event e;

  // Main loop to render the graph
  while (!quit) {
    while (SDL_PollEvent(&e) != 0) {
      if (e.type == SDL_QUIT) 
        quit = true;
    }
    drawBarGraph(renderer, h_data);
  }

  SDL_DestroyRenderer(renderer);
  SDL_DestroyWindow(window);
  SDL_Quit();
  free(d_data);
  free(cpu_data);
  */
#endif
}

template <typename T, fptr_t f>
void test_arrayEquality(T *arr1, T *arr2, int N){
  bool equal = true;
  int index = -1;
  for (int i=0; i<N; i++){
    if (arr1[i] != arr2[i]){
      equal = false;
      index = i;
      break;
    }
  }
  if (!equal)
    printf("NOT EQUAL! DIFFERING VALUES BEGINNING AT INDEX %d!\n", index);
  else
    printf("EQUAL!\n");
}

template <typename T, fptr_t f>
void selection_sort(T* a, int N){
  int temp;
  for (int i=0; i<N; i++){
    for (int j=i+1; j<N; j++){
      if (f(a[j],a[i])){
        temp = a[i];
        a[i] = a[j];
        a[j] = temp;
      }
    }
  }
}
