/*
 * (C) Copyright 2016-2018 Ben Karsin, Nodari Sitchinava
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */


#include<stdio.h>
#include<iostream>
#include<fstream>
#include<vector>
#include<cmath>
#include<random>
#include<algorithm>
#include"multimergesort.hxx"
#include"buildData.h"

#define DEBUG 1  // Set this to 1 to check that the output is correctly sorted
#define PRINT 0  // Set this to 1 to print first M elements of the array for further debugging
#define ITERS 1 // Number of iterations to compute average runtime
#define BLOCKS 128

/* CPU FUNCTION HEADERS*/
template<typename T>
void test_multimergesort(int p, int N);
template<typename T>
void test_squareSort(int N);


int main(int argc, char** argv) {

  if(argc != 2) {
	printf("Usage: ./testMM <N>\n");
	exit(1);
  }

  int N = atoi(argv[1]);
  // test_multimergesort<DATATYPE>(BLOCKS, N);
  test_squareSort<DATATYPE>(N);

  return 0;
}

// Create random data and sort it...
template<typename T>
void test_multimergesort(int p, int N) {

  hipEvent_t start, stop;
  float time_elapsed=0.0;
  float minTime=99999;
  float maxTime=0.0;

 // Create sample sorted lists
  T* h_data = (T*)malloc(N*sizeof(T));

  T* d_data;
  T* d_output;
  hipMalloc(&d_data, N*sizeof(T));
  hipMalloc(&d_output, N*sizeof(T));
  float total_time=0.0;

srand(time(NULL));
for(int it=0; it<ITERS; it++) {

// Create random list to be sorted
  create_random_list<T>(h_data, N, 0);

// Copy list to GPU
  hipMemcpy(d_data, h_data, N*sizeof(T), hipMemcpyHostToDevice);

// Zero out result array
  hipMalloc(&d_output, N*sizeof(T));
//  hipMemset(&d_output, 0, N*sizeof(T));

  hipDeviceSynchronize();
// Timer functions
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

// Run GPU-MMS.  T is datatype and cmp is comparison function (defined in cmp.hxx)
  d_output = multimergesort<T,cmp>(d_data, d_output, h_data, p, N);
  hipDeviceSynchronize();

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_elapsed, start, stop);
  total_time += time_elapsed;
  if(time_elapsed < minTime) minTime = time_elapsed;
  if(time_elapsed > maxTime) maxTime = time_elapsed;
  time_elapsed=0.0;

if(it<ITERS-1)
  hipFree(&d_output);
}
total_time = total_time/ITERS;
printf("%lf %lf %lf\n", total_time, minTime, maxTime);

// copy sorted result back to CPU
  hipMemcpy(h_data, d_output, N*sizeof(T), hipMemcpyDeviceToHost);

// If debug mode is on, check that output is correct
#ifdef DEBUG
  bool error=false;
  for(int i=2; i<N-1; i++) {
    if(host_cmp<int>(h_data[i], h_data[i-1])) {
      error=true;
    }
  }
  if(error)
    printf("NOT SORTED!\n");
  else
    printf("SORTED!\n");
#if PRINT == 1
  printf("[%d", h_data[0]);
  for (int i = 1; i < M; i++)
    printf(", %d", h_data[i]);
  printf("]\n");
#endif
#endif

  hipFree(d_data);
  hipFree(d_output);
  free(h_data);
}


// Function to test just the basecase method
template<typename T>
void test_squareSort(int N) {

  hipEvent_t start, stop;
  float time_elapsed=0.0;
  float total_time=0.0;

  T* h_data = (T*)malloc(N*sizeof(T));
  T* d_data;
  hipMalloc(&d_data, N*sizeof(T));
  srand(time(NULL));

for(int it=0; it < ITERS; it++) {
  create_random_list<T>(h_data, N, 0);

  hipMemcpy(d_data, h_data, N*sizeof(T), hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  squareSort<T, cmp><<<((N/M)/(THREADS/W)),THREADS>>>(d_data, N); // number of blocks was initially BLOCKS, but that was incorrect. squareSort is implemented to have grid dimensions that scale with input size
  hipDeviceSynchronize();
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_elapsed, start, stop);

  total_time += time_elapsed;
}

printf("%lf ", total_time/ITERS);
printf("\n");

  hipMemcpy(h_data, d_data, N*sizeof(T), hipMemcpyDeviceToHost);

  bool sorted=true;

#if PRINT == 1
  printf("[%d", h_data[0]);
  for (int i = 1; i < N; i++)
    printf(", %d", h_data[i]);
  printf("]\n");
#endif

  for(int j=0; j<N; j+=M) {
    for(int i=1; i<M; i++) {
      if(i+j<N && host_cmp<int>(h_data[i+j], h_data[j+i-1]))
        sorted=false;
    } 
  }
  if(!sorted) {
    printf("NOT SORTED\n");
  } else { 
    printf("SORTED!\n");
  }
}
